#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to fill the device array with true values (1)
__global__ void fillTrueValues(bool *array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        array[idx] = true;
    }
}

int main() {
    int arraySize = 1000;
    size_t arrayBytes = arraySize * sizeof(bool);

    // Allocate device memory
    bool *d_array;
    hipMalloc((void**)&d_array, arrayBytes);

    // Set grid and block sizes
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    // Launch the kernel to fill the array with true values
    fillTrueValues<<<gridSize, blockSize>>>(d_array, arraySize);

    // Copy the result back to the host if needed
    bool *h_array = new bool[arraySize];
    hipMemcpy(h_array, d_array, arrayBytes, hipMemcpyDeviceToHost);

    print
    for(int i=0;i<arraySize;i++)
        std::cout<<h_array[i];

    // Clean up
    hipFree(d_array);
    delete[] h_array;

    return 0;
}
