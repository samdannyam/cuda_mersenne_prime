#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to fill the device array with true values (1)
__global__ void fillValues(bool *array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        array[idx] = true;
    }
}

void fillSize(int arraySize){
    size_t arrayBytes = arraySize * sizeof(bool);

    // Allocate device memory
    bool *d_array;
    hipMalloc((void**)&d_array, arrayBytes);

    // Set grid and block sizes
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;

    // Launch the kernel to fill the array with true values
    fillTrueValues<<<gridSize, blockSize>>>(d_array, arraySize);

    // Copy the result back to the host if needed
    bool *h_array = new bool[arraySize];
    hipMemcpy(h_array, d_array, arrayBytes, hipMemcpyDeviceToHost);

    //calculate
    

    // Clean up
    hipFree(d_array);
    delete[] h_array;
}
int main() {
    int max= 100000000;
    for(int i=82589933; i<max; i+=2){
        fillSize(i);
    }

    return 0;
}
